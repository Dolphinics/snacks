#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sisci_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

sci_error_t err;
sci_desc_t sd;
unsigned nodeid;

__global__ void kernel(void* src, void* dst)
{
    *((int*) dst) = *((int*) src);
    *((int*) src) = 0;
}

void server(unsigned segid)
{
    sci_local_segment_t seg;
    SCICreateSegment(sd, &seg, segid, 0x1000, NULL, NULL, 0, &err);

    SCIPrepareSegment(seg, 0, 0, &err);
    SCISetSegmentAvailable(seg, 0, 0, &err);

    sci_map_t m;
    void* ptr = SCIMapLocalSegment(seg, &m, 0, 0x1000, NULL, 0, &err);

    while (1)
    {
        sleep(2);
        printf("%x\n", *((int*) ptr));
        (*((volatile int*) ptr))++;
    }
}


void client(unsigned nodeid, unsigned segid)
{
    sci_remote_segment_t seg;
    SCIConnectSegment(sd, &seg, nodeid, segid, 0, NULL, NULL, SCI_INFINITE_TIMEOUT, 0, &err);

    sci_map_t m;
    volatile void* ptr = SCIMapRemoteSegment(seg, &m, 0, 0x1000, NULL, 0, &err);
    //*((volatile int*) ptr) = 0xdede;

    SCIRegisterPCIeRequester(sd, 0, 1, 0, SCI_FLAG_PCIE_REQUESTER_GLOBAL, &err);
    if (err != SCI_ERR_OK)
    {
        printf("oh noes\n");
    }
    
    hipSetDevice(0);

    fprintf(stderr, "%p\n", (void*) ptr);

    hipError_t cudaerr = hipHostRegister((void*) ptr, 0x1000, hipHostRegisterIoMemory | hipHostRegisterMapped);
    if (cudaerr != hipSuccess)
    {
        fprintf(stderr, "%s\n", hipGetErrorString(cudaerr));
    }

    void* devp;
    cudaerr = hipHostGetDevicePointer(&devp, (void*) ptr, 0);
    if (cudaerr != hipSuccess)
    {
        fprintf(stderr, "%s\n", hipGetErrorString(cudaerr));
    }

    void* devp2;
    hipMalloc(&devp2, sizeof(int));

    kernel<<<1, 1>>>(devp, devp2);

    int value = 0;
    hipMemcpy(&value, devp2, sizeof(int), hipMemcpyDeviceToHost);
    fprintf(stderr, "%x\n", value);

    sleep(5);
}


int main(int argc, char** argv)
{
    unsigned remote_nodeid = 0;
    unsigned remote_segid = 0;
    unsigned local_segid = 0;

    SCIInitialize(0, &err);

    SCIOpen(&sd, 0, &err);

    SCIGetLocalNodeId(0, &nodeid, 0, &err);
    remote_nodeid = nodeid;

    if (argc > 2)
    {
        remote_nodeid = atoi(argv[1]);
        remote_segid = atoi(argv[2]);
    }
    else if (argc > 1)
    {
        local_segid = atoi(argv[1]);
    }
    else
    {
        fprintf(stderr, "Usage: %s <remote node> <remote segment> | %s <local segment>\n", argv[0], argv[0]);
        return 1;
    }

    if (remote_nodeid != nodeid)
    {
        client(remote_nodeid, remote_segid);
    }
    else
    {
        printf("this node: %u, this segment: %u\n", nodeid, local_segid);
        server(local_segid);
    }

    return 0;
}
